#include <cuda/nvarena.cuh>

#include <iostream>

namespace zhetapi {

// Allocate per megabyte
NVArena::NVArena(size_t mb)
{
	size_t bytes = mb << 20;

	using namespace std;
	cout << "Bytes = " << bytes << endl;

	int *pool;
	hipMalloc(&pool, bytes);

	cout << "pool = " << pool << endl;

	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
}

NVArena::~NVArena()
{
	hipFree(__pool);
}

}
