#include "hip/hip_runtime.h"
#include <timer.hpp>
#include <tensor.hpp>
#include <vector.hpp>
#include <cuda/nvarena.cuh>

#include <iostream>

using namespace zhetapi;
using namespace std;

// Simulation class

void io_block()
{
	static std::string input;

	cout << "Blocking (until enter) ";
	getline(cin, input);
}

template <class T>
__global__
void double_kernel(Vector <T> *vptr)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	// size_t incr = blockDim.x;
	if (i < vptr->size())
		vptr->get(i) *= 2;
}

template <class T>
__global__
void add_kernel(Vector <T> *vptr1, Vector <T> *vptr2, Vector <T> *vptr3)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t stride = blockDim.x * gridDim.x;

	while (i < vptr1->size()) {
		vptr3->get(i) = vptr1->get(i) + vptr2->get(i);

		i += stride;
	}
}

int main()
{
	NVArena arena(4096);

	Vector <double> vec1(10, 5);
	Vector <double> vec2(10, 12.4);

	cout << "vec = " << vec1 << endl;

	Vector <double> *hc1 = vec1.cuda_half_copy(&arena);
	Vector <double> *fc1 = hc1->cuda_full_copy(&arena);

	Vector <double> *hc2 = vec2.cuda_half_copy(&arena);
	Vector <double> *fc2 = hc2->cuda_full_copy(&arena);

	double_kernel <<<10, 1>>> (fc1);
	add_kernel <<<10, 1>>> (fc1, fc2, fc1);

	vec1.cuda_read(hc1);
	cout << "post-vec = " << vec1 << endl;

	arena.free(fc1);
	arena.free(fc2);

	delete hc1;
	delete hc2;

	arena.show_mem_map();
}
